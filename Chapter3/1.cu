#include "hip/hip_runtime.h"
#include <iostream>

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 1
#define GRID_SIZE_X 32
#define GRID_SIZE_Y 1

using namespace std;

__global__ void matrix_multiplication(float * a, float * b, float *c, int N, int K) {
    int rowIdx = threadIdx.x;

    for (int j = 0; j < N; j++)
        for(int k = 0; k < K; k++)
            c[rowIdx][j] += a[rowIdx][k]*b[k][j];
}

int main() {
    int M = 3, K = 4, N = 5;

    float * a, * b, * c;
    float * d_a, *d_b, *d_c;

    a = new float[M*K];
    b = new float[K*N];
    c = new float[M*N];


    for (int i = 0; i < M*K; i++)
        a[i] = rand() % 10;
    
    for (int i = 0; i < K*N; i++)
        b[i] = rand() % 10;

    for (int i = 0; i < M*N; i++)
        c[i] = 0;
    
    hipMalloc((void**)&d_a, M * K * sizeof(float));
    hipMalloc((void**)&d_b, K * N * sizeof(float));
    hipMalloc((void**)&d_c, M * N * sizeof(float));

    hipMemcpy(d_a, a, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, K * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, M * N * sizeof(float), hipMemcpyHostToDevice);
    

    dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
    dim3 blocks(GRID_SIZE_X, GRID_SIZE_Y, 1);
    matrix_multiplication<<<threads, blocks>>>(a, b, c, N, K);

    for (int i = 0; i < M; i++){
        for(int j = 0; j < K; j++)
            cout << a[i][j] << " ";
        cout << endl;
    }
    cout << "===================================" << endl;
    for (int i = 0; i < K; i++){
        for(int j = 0; j < N; j++)
            cout << b[i][j] << " ";
        cout << endl;
    }
    cout << "===================================" << endl;
    for (int i = 0; i < M; i++){
        for(int j = 0; j < N; j++)
            cout << c[i][j] << " ";
        cout << endl;
    }
}